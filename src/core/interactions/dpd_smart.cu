#include "dpd_smart.h"
#include <memory>
#include "pairwise_interactions/dpd.h"
#include "pairwise_interactions/smartdpd.h"
#include "pairwise.impl.h"
#include "particle_kernel.h"

#include "pairwise_interactions/FlowProperties.h"


#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>




InteractionFlowProperty::InteractionFlowProperty(const YmrState *state, std::string name, float rc) :
    Interaction(state, name, rc)
{
    PairwiseFlowProperties fp(rc);
    impl = std::make_unique<InteractionPair<PairwiseFlowProperties>> (state, name, rc, fp);
}

InteractionFlowProperty::~InteractionFlowProperty() = default;

void InteractionFlowProperty::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);

    info("Interaction '%s' requires channel '%s' from PVs '%s' and '%s'",
         name.c_str(), ChannelNames::DPDparameters.c_str(), pv1->name.c_str(), pv2->name.c_str());


    pv1->requireDataPerParticle <Vorticity> (ChannelNames::vorticities,ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Vorticity> (ChannelNames::vorticities,ExtraDataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Vorticity> (ChannelNames::vorticities);
    cl2->requireExtraDataPerParticle <Vorticity> (ChannelNames::vorticities);

    pv1->requireDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities,ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities,ExtraDataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities);
    cl2->requireExtraDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities);

    pv1->requireDataPerParticle <Density_Gradient> (ChannelNames::density_gradients,ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Density_Gradient> (ChannelNames::density_gradients,ExtraDataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Density_Gradient> (ChannelNames::density_gradients);
    cl2->requireExtraDataPerParticle <Density_Gradient> (ChannelNames::density_gradients);

    pv1->requireDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients,ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients,ExtraDataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients);
    cl2->requireExtraDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients);

}

std::vector<Interaction::InteractionChannel> InteractionFlowProperty::getIntermediateOutputChannels() const
{
    return {{ChannelNames::vorticities, Interaction::alwaysActive},{ChannelNames::aprox_densities, Interaction::alwaysActive},{ChannelNames::velocity_gradients, Interaction::alwaysActive},{ChannelNames::density_gradients, Interaction::alwaysActive}};
}
std::vector<Interaction::InteractionChannel> InteractionFlowProperty::getFinalOutputChannels() const
{
    return {};
}

void InteractionFlowProperty::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionFlowProperty::halo (ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}





InteractionSmartDPD::InteractionSmartDPD(const YmrState *state, std::string name, std::string parameterName,std::vector<float> weights, float rc, float a, float gamma, float kbt, float power, bool allocateImpl) :
    Interaction(state, name, rc),
    parameterName(parameterName),a(a), gamma(gamma), kbt(kbt), power(power)
{
    if (allocateImpl)
    {
        auto devP = Weights.hostPtr();
        memcpy(devP, &weights[0], weights.size() * sizeof(float));
        Weights.uploadToDevice(0);
        PairwiseSmartDPD dpd(parameterName,rc, a, gamma, kbt, state->dt, power);
        impl = std::make_unique<InteractionPair<PairwiseSmartDPD>> (state,name,rc,dpd);

    }

}

InteractionSmartDPD::InteractionSmartDPD(const YmrState *state,std::string name,std::string parameterName,std::vector<float> weights, float rc, float a, float gamma, float kbt,  float power) :
    InteractionSmartDPD(state,name,parameterName,weights, rc, a, gamma, kbt, power, true)
{}

InteractionSmartDPD::~InteractionSmartDPD() = default;

void InteractionSmartDPD::setPrerequisites(ParticleVector* pv1, ParticleVector* pv2,CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2,cl1,cl2);
    pv1->requireDataPerParticle <DPDparameter> (ChannelNames::DPDparameters,ExtraDataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <DPDparameter> (ChannelNames::DPDparameters,ExtraDataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <DPDparameter> (ChannelNames::DPDparameters);
    cl2->requireExtraDataPerParticle <DPDparameter> (ChannelNames::DPDparameters);

    auto pv1DPDparameter = pv1->local()->extraPerParticle.getData<DPDparameter>(ChannelNames::DPDparameters)->devPtr();
    auto pv2DPDparameter = pv2->local()->extraPerParticle.getData<DPDparameter>(ChannelNames::DPDparameters)->devPtr();
    int nth = 128;
    int np = pv1->local()->size();
    SAFE_KERNEL_LAUNCH(
        copy_kernel,
        getNblocks(np, nth), nth, 0, 0,
        pv1DPDparameter,pv2DPDparameter,np,a,gamma);


}

std::vector<Interaction::InteractionChannel> InteractionSmartDPD::getFinalOutputChannels() const
{
    return impl->getFinalOutputChannels();
}

std::vector<Interaction::InteractionChannel> InteractionSmartDPD::getIntermediateInputChannels() const
{
    return {{ChannelNames::vorticities, Interaction::alwaysActive},{ChannelNames::aprox_densities, Interaction::alwaysActive},{ChannelNames::velocity_gradients, Interaction::alwaysActive},{ChannelNames::density_gradients, Interaction::alwaysActive}};
}

void InteractionSmartDPD::local(ParticleVector* pv1, ParticleVector* pv2,
                             CellList* cl1, CellList* cl2,
                             hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionSmartDPD::halo   (ParticleVector* pv1, ParticleVector* pv2,
                             CellList* cl1, CellList* cl2,
                             hipStream_t stream)
{
    impl->halo   (pv1, pv2, cl1, cl2,stream);
}


void InteractionSmartDPD::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2,
        float a, float gamma, float kbt, float power)
{
    if (a     == Default) a     = this->a;
    if (gamma == Default) gamma = this->gamma;
    if (kbt   == Default) kbt   = this->kbt;
    if (power == Default) power = this->power;


    PairwiseSmartDPD dpd(parameterName,this->rc, a, gamma, kbt, state->dt, power);
    auto ptr = static_cast<InteractionPair<PairwiseSmartDPD>* >(impl.get());

    ptr->setSpecificPair(pv1->name, pv2->name, dpd);
}
