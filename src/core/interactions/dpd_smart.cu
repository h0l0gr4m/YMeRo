#include "dpd_smart.h"
#include <memory>
#include "pairwise_interactions/dpd.h"
#include "pairwise_interactions/smartdpd.h"
#include "pairwise.impl.h"
#include "particle_kernel.h"

#include "pairwise_interactions/FlowProperties.h"
#include "calculations/nninput_kernel.h"
#include "calculations/NNInputs.h"
#include "calculations/NeuralNet_kernel.h"

#include <core/utils/make_unique.h>
#include <core/pvs/particle_vector.h>




InteractionFlowProperty::InteractionFlowProperty(const YmrState *state, std::string name, float rc) :
    Interaction(state, name, rc)
{
    PairwiseFlowProperties fp(rc);
    impl = std::make_unique<InteractionPair<PairwiseFlowProperties>> (state, name, rc, fp);
}

InteractionFlowProperty::~InteractionFlowProperty() = default;

void InteractionFlowProperty::setPrerequisites(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2, cl1, cl2);

    info("Interaction '%s' requires channel '%s' from PVs '%s' and '%s'",
         name.c_str(), ChannelNames::DPDparameters.c_str(), pv1->name.c_str(), pv2->name.c_str());


    pv1->requireDataPerParticle <Vorticity> (ChannelNames::vorticities,DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Vorticity> (ChannelNames::vorticities,DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Vorticity> (ChannelNames::vorticities);
    cl2->requireExtraDataPerParticle <Vorticity> (ChannelNames::vorticities);

    pv1->requireDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities,DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities,DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities);
    cl2->requireExtraDataPerParticle <Aprox_Density> (ChannelNames::aprox_densities);

    pv1->requireDataPerParticle <Density_Gradient> (ChannelNames::density_gradients,DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Density_Gradient> (ChannelNames::density_gradients,DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Density_Gradient> (ChannelNames::density_gradients);
    cl2->requireExtraDataPerParticle <Density_Gradient> (ChannelNames::density_gradients);

    pv1->requireDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients,DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients,DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients);
    cl2->requireExtraDataPerParticle <Velocity_Gradient> (ChannelNames::velocity_gradients);

}

std::vector<Interaction::InteractionChannel> InteractionFlowProperty::getIntermediateOutputChannels() const
{
    return {{ChannelNames::vorticities, Interaction::alwaysActive},{ChannelNames::aprox_densities, Interaction::alwaysActive},{ChannelNames::velocity_gradients, Interaction::alwaysActive},{ChannelNames::density_gradients, Interaction::alwaysActive}};
}
std::vector<Interaction::InteractionChannel> InteractionFlowProperty::getFinalOutputChannels() const
{
    return {};
}

void InteractionFlowProperty::local(ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionFlowProperty::halo (ParticleVector *pv1, ParticleVector *pv2, CellList *cl1, CellList *cl2, hipStream_t stream)
{
    impl->halo(pv1, pv2, cl1, cl2, stream);
}





InteractionSmartDPD::InteractionSmartDPD(const YmrState *state, std::string name, std::string parameterName,std::string NeuralNetType,std::vector<float> weights, float rc,  float kbt, float power, bool allocateImpl) :
    Interaction(state, name, rc),
    parameterName(parameterName),NeuralNetType(NeuralNetType), kbt(kbt), power(power),weights(weights)
{
    if (allocateImpl)
    {
        PairwiseSmartDPD dpd(parameterName,rc, kbt, state->dt, power);
        impl = std::make_unique<InteractionPair<PairwiseSmartDPD>> (state,name,rc,dpd);

    }

}

InteractionSmartDPD::InteractionSmartDPD(const YmrState *state,std::string name,std::string parameterName,std::string NeuralNetType,std::vector<float> weights, float rc, float kbt,  float power) :
    InteractionSmartDPD(state,name,parameterName,NeuralNetType,weights, rc, kbt, power, true)
{}

InteractionSmartDPD::~InteractionSmartDPD() = default;

void InteractionSmartDPD::setPrerequisites(ParticleVector* pv1, ParticleVector* pv2,CellList *cl1, CellList *cl2)
{
    impl->setPrerequisites(pv1, pv2,cl1,cl2);
    pv1->requireDataPerParticle <DPDparameter> (ChannelNames::DPDparameters,DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <DPDparameter> (ChannelNames::DPDparameters,DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <DPDparameter> (ChannelNames::DPDparameters);
    cl2->requireExtraDataPerParticle <DPDparameter> (ChannelNames::DPDparameters);

    pv1->requireDataPerParticle <NNInput> (ChannelNames::NNInputs,DataManager::PersistenceMode::None);
    pv2->requireDataPerParticle <NNInput> (ChannelNames::NNInputs,DataManager::PersistenceMode::None);

    cl1->requireExtraDataPerParticle <NNInput> (ChannelNames::NNInputs);
    cl2->requireExtraDataPerParticle <NNInput> (ChannelNames::NNInputs);

    if(NeuralNetType=="pseudolinear" || NeuralNetType=="linear")
	{
	    Weights.resize_anew(22);
	    auto hostPtr = Weights.hostPtr();
	    memcpy(hostPtr, &weights[0], weights.size() * sizeof(float));
	    Weights.uploadToDevice(0);
	}
    else if(NeuralNetType=="nonlinear")
        {
 	    Weights.resize_anew(103);
            auto hostPtr = Weights.hostPtr();
            memcpy(hostPtr, &weights[0], weights.size() * sizeof(float));
            Weights.uploadToDevice(0);
	    pv1->requireDataPerParticle <Intermediate_Input> (ChannelNames::Intermediate_Inputs,DataManager::PersistenceMode::None);
	    pv2->requireDataPerParticle <Intermediate_Input> (ChannelNames::Intermediate_Inputs,DataManager::PersistenceMode::None);

	    cl1->requireExtraDataPerParticle <Intermediate_Input> (ChannelNames::Intermediate_Inputs);
	    cl2->requireExtraDataPerParticle <Intermediate_Input> (ChannelNames::Intermediate_Inputs);
	   
 	}  
}


void InteractionSmartDPD::localNeuralNetwork (ParticleVector* pv, CellList* cl,hipStream_t stream)
{
    NNInput_Computation nninputs(rc);
    using ViewType = typename NNInput_Computation::ViewType;
    ViewType view(pv,pv->local());
    int size = view.size;
    nninputs.setup(pv->local());
    int nth = 128;
    SAFE_KERNEL_LAUNCH(
      computeNNInputs,getNblocks(size,nth),nth,0,stream,
      view,nninputs);


    auto pvDPDparameter = pv->local()->dataPerParticle.getData<DPDparameter>(ChannelNames::DPDparameters)->devPtr();
    auto pvNNInputs = pv->local()->dataPerParticle.getData<NNInput>(ChannelNames::NNInputs)->devPtr();
    auto Weights_ptr = Weights.devPtr();
    if(NeuralNetType=="pseudolinear")
	{
				SAFE_KERNEL_LAUNCH(
				PseudolinearNeuralNet,getNblocks(32*size,nth),nth,0,stream,
				size,pvDPDparameter,pvNNInputs,Weights_ptr
				);
	}   
    else if (NeuralNetType =="linear")
	{
				SAFE_KERNEL_LAUNCH(
				LinearNeuralNet,getNblocks(32*size,nth),nth,0,stream,
				size,pvDPDparameter,pvNNInputs,Weights_ptr
				);
	}   

    else if (NeuralNetType == "nonlinear")
	{
  				auto pvIntermediate_Inputs = pv->halo()->dataPerParticle.getData<Intermediate_Input>(ChannelNames::Intermediate_Inputs)->devPtr();
				SAFE_KERNEL_LAUNCH(
				NonLinearNeuralNet_1,getNblocks(4*32*size,nth),nth,0,stream,
				size,pvIntermediate_Inputs,pvNNInputs,Weights_ptr
				);
                                SAFE_KERNEL_LAUNCH(
				NonLinearNeuralNet_2,getNblocks(32*size,nth),nth,0,stream,
				size,pvIntermediate_Inputs,pvDPDparameter,Weights_ptr
				);


	}
   
}

void InteractionSmartDPD::haloNeuralNetwork(ParticleVector* pv,CellList *cl, hipStream_t stream)
{
  NNInput_Computation nninputs(rc);
  using ViewType = typename NNInput_Computation::ViewType;
  ViewType  view (pv,pv->halo());
  int size =view.size;
  nninputs.setup(pv->halo());
  int nth = 128;
  SAFE_KERNEL_LAUNCH(
    computeNNInputs,getNblocks(size,nth),nth,0,stream,
    view,nninputs);


  auto pvDPDparameter = pv->halo()->dataPerParticle.getData<DPDparameter>(ChannelNames::DPDparameters)->devPtr();
  auto pvNNInputs = pv->halo()->dataPerParticle.getData<NNInput>(ChannelNames::NNInputs)->devPtr();
  auto Weights_ptr = Weights.devPtr();
  if(NeuralNetType=="linear")
	{
				SAFE_KERNEL_LAUNCH(
				LinearNeuralNet,getNblocks(32*size,nth),nth,0,stream,
				size,pvDPDparameter,pvNNInputs,Weights_ptr
				);
	}	
  else if(NeuralNetType=="pseudolinear")
         {
                                 SAFE_KERNEL_LAUNCH(
                                 PseudolinearNeuralNet,getNblocks(32*size,nth),nth,0,stream,
                                 size,pvDPDparameter,pvNNInputs,Weights_ptr
                                 );
         }

  else if (NeuralNetType =="nonlinear")
	{
  				auto pvIntermediate_Inputs = pv->halo()->dataPerParticle.getData<Intermediate_Input>(ChannelNames::Intermediate_Inputs)->devPtr();
				SAFE_KERNEL_LAUNCH(
				NonLinearNeuralNet_1,getNblocks(4*32*size,nth),nth,0,stream,
				size,pvIntermediate_Inputs,pvNNInputs,Weights_ptr
				);
                                SAFE_KERNEL_LAUNCH(
				NonLinearNeuralNet_2,getNblocks(32*size,nth),nth,0,stream,
				size,pvIntermediate_Inputs,pvDPDparameter,Weights_ptr
				);

	}
}


std::vector<Interaction::InteractionChannel> InteractionSmartDPD::getFinalOutputChannels() const
{
    return impl->getFinalOutputChannels();
}

std::vector<Interaction::InteractionChannel> InteractionSmartDPD::getIntermediateInputChannels() const
{
    return {{ChannelNames::vorticities, Interaction::alwaysActive},{ChannelNames::aprox_densities, Interaction::alwaysActive},{ChannelNames::velocity_gradients, Interaction::alwaysActive},{ChannelNames::density_gradients, Interaction::alwaysActive}};
}

void InteractionSmartDPD::local(ParticleVector* pv1, ParticleVector* pv2,
                             CellList* cl1, CellList* cl2,
                             hipStream_t stream)
{
    impl->local(pv1, pv2, cl1, cl2, stream);
}

void InteractionSmartDPD::halo   (ParticleVector* pv1, ParticleVector* pv2,
                             CellList* cl1, CellList* cl2,
                             hipStream_t stream)
{
    impl->halo   (pv1, pv2, cl1, cl2,stream);
}


void InteractionSmartDPD::setSpecificPair(ParticleVector* pv1, ParticleVector* pv2,
         float kbt, float power)
{
    if (kbt   == Default) kbt   = this->kbt;
    if (power == Default) power = this->power;


    PairwiseSmartDPD dpd(parameterName,this->rc, kbt, state->dt, power);
    auto ptr = static_cast<InteractionPair<PairwiseSmartDPD>* >(impl.get());

    ptr->setSpecificPair(pv1->name, pv2->name, dpd);
}
